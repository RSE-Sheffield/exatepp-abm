#include "hip/hip_runtime.h"
#include "exateppabm/population.h"

#include <fmt/core.h>

#include <algorithm>
#include <numeric>
#include <memory>
#include <random>
#include <vector>

#include "exateppabm/demographics.h"
#include "exateppabm/disease.h"
#include "exateppabm/person.h"
#include "exateppabm/input.h"

namespace exateppabm {
namespace population {

namespace {

// File-scoped array  contianing the number of infected agents per demographic from population initialisation. This needs to be made accessible to a FLAME GPU Init func due to macro environment property limitations.

std::array<std::uint64_t, demographics::AGE_COUNT> infectedPerDemographic = {};


/**
 * in-place inclusive scan, for libstdc++ which does not support c++17 (i.e. GCC 8)
 */
template <typename T>
void inplace_inclusive_scan(T& container) {
    // @todo - use cmake to detect which path needs taking
    // std::inclusive_scan(container.begin(), container.end(), container.begin());
    // return;
    // @todo - refactor this into a testable method, in a util namespace?
    if (container.size() <= 1) {
        return;
    }
    // Naive in-place inclusive scan for libstc++8
    for (size_t i = 1; i < container.size(); ++i) {
        container[i] = container[i - 1] + container[i];
    }
}

}  // namespace

std::unique_ptr<flamegpu::AgentVector> generate(flamegpu::ModelDescription& model, const exateppabm::input::config config, const bool verbose, const float env_width, const float interactionRadius) {
    fmt::print("@todo - validate config inputs when generated agents (pop size, initial infected count etc)\n");

    // @todo - assert that the requested initial population is non zero.
    auto pop = std::make_unique<flamegpu::AgentVector>(model.Agent(exateppabm::person::NAME), config.n_total);

    std::uint64_t sq_width = static_cast<std::uint64_t>(env_width);
    // float expectedNeighbours = interactionRadius * interactionRadius;
    // fmt::print("sq_width {} interactionRadius {} expectedNeighbours {}\n", sq_width, interactionRadius, expectedNeighbours);

    // seed host rng for population generation.
    // @todo - does this want to be a separate seed from the config file?
    std::mt19937_64 rng(config.rng_seed);


    // Need to initialise a fixed number of individuals as infected.
    // This not very scalable way of doing it, is to create a vector with one element per individual in the simulation, initialised to false
    // set the first n_seed_infection elements to true/1
    // Shuffle the vector,  and query at agent creation time
    // RNG sampling in-loop would be more memory efficient, but harder to guarantee that exactly enough are created. This will likely be replaced anyway, so quick and dirty is fine.
    std::vector<bool> infected_vector(config.n_total);
    std::fill(infected_vector.begin(), infected_vector.begin() + std::min(config.n_total, config.n_seed_infection), true);
    std::shuffle(infected_vector.begin(), infected_vector.end(), rng);

    // Prepare a probability matrix for selecting an age demographic for the agent based on the ratio from the configuration.
    // @todo abstract this into class/methods.
    // @todo - this hardcoded 9 is a bit grim. Maybe enums can help?
    std::uint64_t configDemographicSum = config.population_0_9 + config.population_10_19 + config.population_20_29 + config.population_30_39 + config.population_40_49 + config.population_50_59 + config.population_60_69 + config.population_70_79 + config.population_80;
    // @todo - map might be more readable than an array (incase the underlying class enum values are ever changed to be a different order?)
    std::array<float, demographics::AGE_COUNT> demographicProbabilties =  {{
        config.population_0_9 / static_cast<float>(configDemographicSum),
        config.population_10_19 / static_cast<float>(configDemographicSum),
        config.population_20_29 / static_cast<float>(configDemographicSum),
        config.population_30_39 / static_cast<float>(configDemographicSum),
        config.population_40_49 / static_cast<float>(configDemographicSum),
        config.population_50_59 / static_cast<float>(configDemographicSum),
        config.population_60_69 / static_cast<float>(configDemographicSum),
        config.population_70_79 / static_cast<float>(configDemographicSum),
        config.population_80 / static_cast<float>(configDemographicSum)
    }};
    // Perform an inclusive scan to convert to cumulative probability
    // Using a local method which supports inclusive scans in old libstc++
    inplace_inclusive_scan(demographicProbabilties);
    // std::inclusive_scan(demographicProbabilties.begin(), demographicProbabilties.end(), demographicProbabilties.begin());
    std::array<demographics::Age, demographics::AGE_COUNT> allDemographics = {{
        demographics::Age::AGE_0_9,
        demographics::Age::AGE_10_19,
        demographics::Age::AGE_20_29,
        demographics::Age::AGE_30_39,
        demographics::Age::AGE_40_49,
        demographics::Age::AGE_50_59,
        demographics::Age::AGE_60_69,
        demographics::Age::AGE_70_79,
        demographics::Age::AGE_80
    }};

    // per demo total is not an output in time series.
    // Alternately, we need to initialise the exact number of each age band, not RNG, and just scale it down accordingly. Will look at in "realistic" population generation
    std::array<std::uint64_t, demographics::AGE_COUNT> createdPerDemographic = {{0, 0, 0, 0, 0, 0, 0, 0, 0}};
    // reset per demographic count of the number initialised agents in each infection state.
    infectedPerDemographic = {{0, 0, 0, 0, 0, 0, 0, 0, 0}};

    std::uniform_real_distribution<float> demo_dist(0.0f, 1.0f);

    unsigned idx = 0;
    for (auto person : *pop) {
        // Infections status
        disease::SEIR::InfectionState infectionStatus = infected_vector.at(idx) ? disease::SEIR::InfectionState::Infected : disease::SEIR::InfectionState::Susceptible;
        person.setVariable<disease::SEIR::InfectionStateUnderlyingType>(exateppabm::person::v::INFECTION_STATE, infectionStatus);

        // Demographic
        // @todo - this is a bit grim, enum class aren't as nice as hoped.
        float demo_random = demo_dist(rng);
        // @todo - abstract this into a method.
        demographics::Age demo = demographics::Age::AGE_0_9;
        for (demographics::AgeUnderlyingType i = 0; i < demographics::AGE_COUNT; i++) {
            if (demo_random < demographicProbabilties[i]) {
                demo = allDemographics[i];
                createdPerDemographic[i]++;
                if (infectionStatus == disease::SEIR::Infected) {
                    infectedPerDemographic[i]++;
                }
                break;
            }
        }
        person.setVariable<demographics::AgeUnderlyingType>(exateppabm::person::v::AGE_DEMOGRAPHIC, demo);

        // Location in 3D space (temp/vis)
        unsigned row = idx / sq_width;
        unsigned col = idx % sq_width;
        person.setVariable<float>(exateppabm::person::v::x, col);  // @todo temp
        person.setVariable<float>(exateppabm::person::v::y, row);  // @todo -temp
        person.setVariable<float>(exateppabm::person::v::z, 0);  // @todo -temp

        // Inc counter
        ++idx;
    }

    if (verbose) {
        // Print a summary of population creation for now.
        fmt::print("Created {} people with {} infected.\n", config.n_total, config.n_seed_infection);
        fmt::print("Demographics {{\n");
        fmt::print("   0- 9 = {}\n", createdPerDemographic[0]);
        fmt::print("  10-19 = {}\n", createdPerDemographic[1]);
        fmt::print("  20-29 = {}\n", createdPerDemographic[2]);
        fmt::print("  30-39 = {}\n", createdPerDemographic[3]);
        fmt::print("  40-49 = {}\n", createdPerDemographic[4]);
        fmt::print("  50-59 = {}\n", createdPerDemographic[5]);
        fmt::print("  60-69 = {}\n", createdPerDemographic[6]);
        fmt::print("  70-79 = {}\n", createdPerDemographic[7]);
        fmt::print("  80+   = {}\n", createdPerDemographic[8]);
        fmt::print("}}\n");
    }

    return pop;
}

std::array<std::uint64_t, demographics::AGE_COUNT> getPerDemographicInitialInfectionCount() {
    return infectedPerDemographic;
}

}  // namespace population
}  // namespace exateppabm
