#include "exateppabm/util.h"

#include <hip/hip_runtime.h>

#include <cstdint>
#include <string>

#include "fmt/core.h"

namespace exateppabm {
namespace util {

std::string getGPUName(int ordinal) {
    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop, ordinal) == hipSuccess) {
        return std::string(prop.name);
    } else {
        return "unknown";
    }
}

std::uint32_t getGPUMultiProcessorCount(int ordinal) {
    int value = 0;
    // don't check the error, if the ordinal is bad it will be caught elsewhere
    hipDeviceGetAttribute(&value, hipDeviceAttributeMultiprocessorCount, ordinal);
    return value;
}

std::size_t getGPUMemory(int ordinal) {
    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop, ordinal) == hipSuccess) {
        return prop.totalGlobalMem;
    } else {
        return 0;
    }
}

void initialiseCUDAContext(int ordinal) {
    int device_count = 0;
    // don't report any errors, they will be found later nicely.
    if (hipGetDeviceCount(&device_count) == hipSuccess) {
        if (hipSetDevice(ordinal) == hipSuccess) {
            hipFree(nullptr);
        }
    }
}

bool getSeatbeltsEnabled() {
#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
    return true;
#else  // !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
    return false;
#endif  // !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
}

std::string getCMakeBuildType() {
#if defined(CMAKE_BUILD_TYPE)
    return CMAKE_BUILD_TYPE;
#else  // defined(CMAKE_BUILD_TYPE)
    return "";
#endif  // defined(CMAKE_BUILD_TYPE)
}

}  // namespace util
}  // namespace exateppabm
