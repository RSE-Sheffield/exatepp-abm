#include "hip/hip_runtime.h"
#include "exateppabm/disease/SEIR.h"

#include "exateppabm/demographics.h"
#include "exateppabm/person.h"

namespace exateppabm {
namespace disease {
namespace SEIR {

/**
 * FLAME GPU Agent function which progresses the disease of the current agent.
 *
 * @todo -include agent demographics in this?
 * @todo - env based parameters?
 */
FLAMEGPU_AGENT_FUNCTION(progressDisease, flamegpu::MessageNone, flamegpu::MessageNone) {
    // Get the current timestep / day
    std::uint32_t today = FLAMEGPU->getStepCounter();

    // Get the current agents infection status
    auto infectionState = FLAMEGPU->getVariable<disease::SEIR::InfectionStateUnderlyingType>(person::v::INFECTION_STATE);
    // Get the time they last changed state
    std::uint32_t dayOfLastStateChange = FLAMEGPU->getVariable<std::uint32_t>(person::v::INFECTION_STATE_CHANGE_DAY);
    // Get the duration of their state current state
    float stateDuration = FLAMEGPU->getVariable<float>(person::v::INFECTION_STATE_DURATION);
    // Ready to change state if today is past the next scheduled state change
    bool readyToChange = today >= dayOfLastStateChange + std::ceil(stateDuration);
    // For each different initial state, change if required and compute the next state's duration.
    if (infectionState == disease::SEIR::InfectionState::Exposed) {
        // Exposed to Infected, if enough time has passed
        if (readyToChange) {
            exposedToInfected(FLAMEGPU, infectionState);
        }
    } else if (infectionState == disease::SEIR::InfectionState::Infected) {
        // Infected to Recovered if enough time has passed
        if (readyToChange) {
            infectedToRecovered(FLAMEGPU, infectionState);
        }
    } else if (infectionState == disease::SEIR::InfectionState::Recovered) {
        // Recovered to Susceptible, if enough time has passed.
        if (readyToChange) {
            recoveredToSusceptible(FLAMEGPU, infectionState);
        }
    }
    return flamegpu::ALIVE;
}

void define(flamegpu::ModelDescription& model, const exateppabm::input::config& params) {
    // Get a handle to the model environment description object
    flamegpu::EnvironmentDescription env = model.Environment();

    // Add a macro environment property (Atomically mutable) for tracking the cumulative number of infected individuals in each demographic.
    // @todo - should this be defined in time series instead? as its data collection not behaviour?
    env.newMacroProperty<std::uint32_t, demographics::AGE_COUNT>("total_infected_per_demographic");

    // Add a number of model parameters to the environment, initialised with the value from the configuration file
    // @todo - not all of these feel right here / add constexpr strings somewhere.
    env.newProperty<float>("mean_time_to_infected", params.mean_time_to_infected);
    env.newProperty<float>("sd_time_to_infected", params.sd_time_to_infected);
    env.newProperty<float>("mean_time_to_recovered", params.mean_time_to_recovered);
    env.newProperty<float>("sd_time_to_recovered", params.sd_time_to_recovered);
    env.newProperty<float>("mean_time_to_susceptible", params.mean_time_to_susceptible);
    env.newProperty<float>("sd_time_to_susceptible", params.sd_time_to_susceptible);

    // Get a handle for the Person agent type
    flamegpu::AgentDescription person = model.Agent(exateppabm::person::NAME);

    // @todo - add agent variables here? Need to decide where is best, depending on how things are used / accessed?

    // Define disease-only agent functions and message lists
    // @todo - consider using FLAME GPU 2 states along with the DiseaseState to improve performance if possible?
    // @todo - add function name to a namespace
    flamegpu::AgentFunctionDescription diseaseProgression = person.newFunction("progressDisease", progressDisease);
    diseaseProgression.setInitialState(exateppabm::person::states::DEFAULT);
    diseaseProgression.setEndState(exateppabm::person::states::DEFAULT);
}

void appendLayers(flamegpu::ModelDescription& model) {
    // Add the disease progression function for person agents
    {
        auto layer = model.newLayer();
        layer.addAgentFunction(exateppabm::person::NAME, "progressDisease");
    }
}

}  // namespace SEIR
}  // namespace disease
}  // namespace exateppabm
